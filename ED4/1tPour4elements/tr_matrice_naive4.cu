#include "hip/hip_runtime.h"
#include "../matrice.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define DIM_PORTION 32
#define LIGNES_BLOC 8

// Code GPU

__global__ void transpose_device(const float *input, float *output, int n)
{
	int largeur_matrice = blockDim.x * gridDim.x;
	int x_matrice = blockIdx.x * blockDim.x + threadIdx.x;
	int y_matrice = blockIdx.y * blockDim.y + threadIdx.y;
	int indice_lin = 0;
	for (int j = 0; j < DIM_PORTION; j += LIGNES_BLOC)
	{
		indice_lin = (largeur_matrice * (y_matrice + j)) + x_matrice; // addresse

		if (x_matrice < n && (y_matrice + j) < n) //j ou pas?
		{

			output[((x_matrice)*n) + y_matrice + j] = input[((y_matrice + j) * n) + x_matrice];
		}
		else
		{
			return;
		}
	}
}

// Code CPU


int main(int argc, char **argv)
{
	int n(0);
	bool affiche(false);
		user_input(affiche,n,argc,argv);


	size_t size = n * n * sizeof(float);

	// Matrices CPU
	float *h_A = nullptr, *h_B = nullptr;
	// Matrices GPU
	float *d_A = nullptr, *d_B = nullptr;

	// Allocatation des vecteurs dans la mémoire CPU
	h_A = new float[n * n];
	h_B = new float[n * n];

	// Allocation des vecteurs dans la mémoire GPU
	checkCudaErrors(hipMalloc((void **)&d_A, size));
	checkCudaErrors(hipMalloc((void **)&d_B, size));

	// Initialisation de la matrice A
	srand(time(NULL));
	genmat(h_A, n);

	// Copie de la matrice A dans la mémoire GPU
	checkCudaErrors(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

	// Definition de la taille des blocs et de la grille
	dim3 threadsPerBlock(DIM_PORTION, DIM_PORTION);
	dim3 numBlocks(ceil(n / (float)threadsPerBlock.x), ceil(n / (float)threadsPerBlock.x));
	std::cout << "bx: " << numBlocks.x << " by: " << numBlocks.y << "\n";

	transpose_device<<<numBlocks, threadsPerBlock>>>(d_A, d_B, n);
	checkCudaErrors(hipPeekAtLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// Copie du résultat
	checkCudaErrors(hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost));
	printf("Erreur max: %e\n", verify(h_A, h_B, n));

	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	const int nb = 10;
	checkCudaErrors(hipEventRecord(start, 0));
	for (int i = 0; i < nb; i++)
		transpose_device<<<numBlocks, threadsPerBlock>>>(d_A, d_B, n);
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	float t_ms;
	checkCudaErrors(hipEventElapsedTime(&t_ms, start, stop));
	t_ms /= nb;
	t_ms /= 1000;
	float octets_echanges(2 * size / pow(10, 9));

	affichage_resultats_du_kernel(h_A, h_B, n, t_ms, octets_echanges, affiche);

	return 0;
}
