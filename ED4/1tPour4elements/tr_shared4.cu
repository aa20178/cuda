#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define DIM_PORTION 32
#define LIGNES_BLOC 8

// Code GPU

__global__ void transpose_device(const float *input, float *output, int n)
{
	__shared__ float matrice_shared[DIM_PORTION][DIM_PORTION];

	int largeur_matrice = blockDim.x * gridDim.x;
	int x_matrice = blockIdx.x * blockDim.x + threadIdx.x;
	int y_matrice = blockIdx.y * blockDim.y + threadIdx.y;
	int indice_lin = 0;

	for (int j = 0; j < DIM_PORTION; j += LIGNES_BLOC)
	{
		if (x_matrice < n && y_matrice < n)
		{
			matrice_shared[threadIdx.y][threadIdx.x] = input[(j + y_matrice) * n + x_matrice];
		}

		__syncthreads();

		if (x_matrice < n && y_matrice < n)
		{
			output[(y_matrice + j) * n + x_matrice] = matrice_shared[threadIdx.x][threadIdx.y];
		}
	}
}

// Code CPU
void afficher_matrice(float *A, int n)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			std::cout << A[i * n + j] << "  ";
		}
		std::cout << std::endl;
	}
}

void genmat(float *A, int n)
{
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			A[i * n + j] = rand() / (float)RAND_MAX;
}
float verify(const float *A, const float *B, int n)
{
	float error = 0;
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			error = std::max(error, abs(A[i * n + j] - B[i * n + j]));

	return error;
}

int compter_occurences_degalite(float *h_A, float *h_B, int n) // n c'est le côté de la mat
{
	int compteur = 0;

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			if (h_A[i * n + j] == h_B[i * n + j])
			{
				compteur++;
			}
		}
	}
	return compteur;
}
int compter_occurences_de_difference(float *h_A, float *h_B, int n) // n c'est le côté de la mat
{
	int compteur = 0;

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			if (h_A[i * n + j] == h_B[i * n + j])
			{
				compteur++;
			}
		}
	}
	return compteur;
}
int main(int argc, char **argv)
{

	int n = 0;
	bool affiche(false);

	if (argc < 2)
	{
		std::cout << argc << " il faut entrer un argument (taille matrice) " << std::endl;
		exit(-1);
	}
	if (argv[1] != NULL && atoi(argv[1]) > 1)
	{
		n = atoi(argv[1]);
	}
	if (argv[2] != NULL)
	{
		affiche = true;
	}

	size_t size = n * n * sizeof(float);
	// Matrices CPU
	float *h_A = nullptr, *h_B = nullptr;
	// Matrices GPU
	float *d_A = nullptr, *d_B = nullptr;

	// Allocatation des vecteurs dans la mémoire CPU
	h_A = new float[n * n];
	h_B = new float[n * n];

	// Allocation des vecteurs dans la mémoire GPU
	checkCudaErrors(hipMalloc((void **)&d_A, size));
	checkCudaErrors(hipMalloc((void **)&d_B, size));

	// Initialisation de la matrice A
	srand(time(NULL));
	genmat(h_A, n);

	// Copie de la matrice A dans la mémoire GPU
	checkCudaErrors(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

	// Definition de la taille des blocs et de la grille
	dim3 threadsPerBlock(DIM_PORTION, DIM_PORTION);
	dim3 numBlocks(ceil(n / (float)threadsPerBlock.x), ceil(n / (float)threadsPerBlock.x));
	std::cout << "bx: " << numBlocks.x << " by: " << numBlocks.y << "\n";

	transpose_device<<<numBlocks, threadsPerBlock>>>(d_A, d_B, n);
	checkCudaErrors(hipPeekAtLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// Copie du résultat
	checkCudaErrors(hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost));
	printf("Erreur max: %e\n", verify(h_A, h_B, n));

	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	const int nb = 10;
	checkCudaErrors(hipEventRecord(start, 0));
	for (int i = 0; i < nb; i++)
		transpose_device<<<numBlocks, threadsPerBlock>>>(d_A, d_B, n);
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	float t_ms;
	checkCudaErrors(hipEventElapsedTime(&t_ms, start, stop));
	t_ms /= nb;
	t_ms /= 1000;
	float octets_echanges(2 * size / pow(10, 9));

	printf("Temps d'exécution du Kernel : %e (ms)\n", t_ms);
	printf("Bande passante GPU: %e GO/s\n", octets_echanges / t_ms);

	if (affiche == true)
	{

		std::cout << " A : " << std::endl;
		afficher_matrice(h_A, n);

		std::cout << " B : " << std::endl;
		afficher_matrice(h_B, n);
	}
	std::cout << " nombre d'éléments inchangés (transposée VS originale)  : " << compter_occurences_degalite(h_A, h_B, n) << " et on devrait en avoir " << n <<" (taille de la diagonale). " <<std::endl;

	if (d_A)
		hipFree(d_A);
	if (d_B)
		hipFree(d_B);

	// Deallocation de la memoire CPU
	if (h_A)
		delete[] h_A;
	if (h_B)
		delete[] h_B;
	return 0;
}
