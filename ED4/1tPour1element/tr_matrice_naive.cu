#include "hip/hip_runtime.h"
#include "../matrice.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define DIM_PORTION 32

// Code GPU

__global__ void copymat_device(const float *input, float *output, int n)
{
	int x_matrice = blockIdx.x * blockDim.x + threadIdx.x;
	int y_matrice = blockIdx.y * blockDim.y + threadIdx.y;

	if (x_matrice < n && y_matrice < n)
	{
		output[(x_matrice * n) + y_matrice] = input[(y_matrice * n) + x_matrice];
	}
}

int main(int argc, char **argv)
{
	int n(0);
	bool affiche(false);
	user_input(affiche,n,argc,argv);

	size_t size = n * n * sizeof(float);
	// Matrices CPU
	float *h_A = nullptr, *h_B = nullptr, *h_B_by_cpu = nullptr;
	// Matrices GPU
	float *d_A = nullptr, *d_B = nullptr;

	// Allocatation des vecteurs dans la mémoire CPU
	h_A = new float[n * n];
	h_B = new float[n * n];
	h_B_by_cpu = new float[n * n];

	// Allocation des vecteurs dans la mémoire GPU
	checkCudaErrors(hipMalloc((void **)&d_A, size));
	checkCudaErrors(hipMalloc((void **)&d_B, size));

	// Initialisation de la matrice A
	srand(time(NULL));
	genmat(h_A, n);

	// Copie de la matrice A dans la mémoire GPU
	checkCudaErrors(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

	// Definition de la taille des blocs et de la grille
	dim3 threadsPerBlock(DIM_PORTION, DIM_PORTION);
	dim3 numBlocks(ceil(n / (float)threadsPerBlock.x), ceil(n / (float)threadsPerBlock.x));
	std::cout << "bx: " << numBlocks.x << " by: " << numBlocks.y << "\n";

	copymat_device<<<numBlocks, threadsPerBlock>>>(d_A, d_B, n);
	checkCudaErrors(hipPeekAtLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// Copie du résultat
	checkCudaErrors(hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost));
	printf("Erreur max: %e\n", verify(h_A, h_B, n));

	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	const int nb = 10;
	checkCudaErrors(hipEventRecord(start, 0));
	for (int i = 0; i < nb; i++)
		copymat_device<<<numBlocks, threadsPerBlock>>>(d_A, d_B, n);
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	float t_ms;
	checkCudaErrors(hipEventElapsedTime(&t_ms, start, stop));
	t_ms /= nb;
	t_ms /= 1000;
	float octets_echanges(2 * size / pow(10, 9));

	affichage_resultats_du_kernel(h_A, h_B, n, t_ms, octets_echanges, affiche);

	return 0;
}
